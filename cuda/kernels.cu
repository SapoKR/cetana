#include "hip/hip_runtime.h"
#include "gcc13_compat.h"

extern "C" __global__ void vector_add_kernel(float *result, const float *a, const float *b, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        result[idx] = a[idx] + b[idx];
    }
}

extern "C" __global__ void vector_subtract_kernel(float *result, const float *a, const float *b, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        result[idx] = a[idx] - b[idx];
    }
}

extern "C" __global__ void vector_multiply_kernel(float *result, const float *a, const float *b, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        result[idx] = a[idx] * b[idx];
    }
}

extern "C" __global__ void vector_divide_kernel(float *result, const float *a, const float *b, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        result[idx] = a[idx] / b[idx];
    }
}

extern "C" __global__ void vector_exp_kernel(float *result, const float *a, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        result[idx] = expf(a[idx]);
    }
}

extern "C" __global__ void vector_log_kernel(float *result, const float *a, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        result[idx] = logf(a[idx]);
    }
}

extern "C" __global__ void vector_pow_kernel(float *result, const float *a, float power, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        result[idx] = powf(a[idx], power);
    }
}

extern "C" __global__ void vector_sqrt_kernel(float *result, const float *a, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        result[idx] = sqrtf(a[idx]);
    }
}

extern "C" __global__ void matrix_multiply_kernel(float *result, const float *a, const float *b,
                                                  int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k)
    {
        float sum = 0.0f;
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        result[row * k + col] = sum;
    }
}

extern "C" __global__ void reduce_sum_kernel(float* output, const float* input, int n) {
    extern __shared__ float sdata[];
    
    // Each thread loads one element from global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory, handling out-of-bounds
    sdata[tid] = (i < n) ? input[i] : 0.0f;
    __syncthreads();
    
    // Do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    // Write result for this block to global memory
    if (tid == 0) output[blockIdx.x] = sdata[0];
}

extern "C" __global__ void reduce_blocks_kernel(float* output, const float* input, int n) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    
    // Load data directly to shared memory
    sdata[tid] = (tid < n) ? input[tid] : 0.0f;
    __syncthreads();
    
    // Do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    // Write result to output
    if (tid == 0) *output = sdata[0];
}

// Kernel launch wrapper functions
extern "C" int launch_vector_add_kernel(
    float *result, const float *a, const float *b, int n,
    unsigned int grid_dim_x, unsigned int grid_dim_y, unsigned int grid_dim_z,
    unsigned int block_dim_x, unsigned int block_dim_y, unsigned int block_dim_z,
    unsigned int shared_mem_bytes, void *stream)
{
    dim3 grid(grid_dim_x, grid_dim_y, grid_dim_z);
    dim3 block(block_dim_x, block_dim_y, block_dim_z);
    
    vector_add_kernel<<<grid, block, shared_mem_bytes, (hipStream_t)stream>>>(result, a, b, n);
    return hipGetLastError();
}

extern "C" int launch_vector_multiply_kernel(
    float *result, const float *a, const float *b, int n,
    unsigned int grid_dim_x, unsigned int grid_dim_y, unsigned int grid_dim_z,
    unsigned int block_dim_x, unsigned int block_dim_y, unsigned int block_dim_z,
    unsigned int shared_mem_bytes, void *stream)
{
    dim3 grid(grid_dim_x, grid_dim_y, grid_dim_z);
    dim3 block(block_dim_x, block_dim_y, block_dim_z);
    
    vector_multiply_kernel<<<grid, block, shared_mem_bytes, (hipStream_t)stream>>>(result, a, b, n);
    return hipGetLastError();
}

extern "C" int launch_vector_subtract_kernel(
    float *result, const float *a, const float *b, int n,
    unsigned int grid_dim_x, unsigned int grid_dim_y, unsigned int grid_dim_z,
    unsigned int block_dim_x, unsigned int block_dim_y, unsigned int block_dim_z,
    unsigned int shared_mem_bytes, void *stream)
{
    dim3 grid(grid_dim_x, grid_dim_y, grid_dim_z);
    dim3 block(block_dim_x, block_dim_y, block_dim_z);
    
    vector_subtract_kernel<<<grid, block, shared_mem_bytes, (hipStream_t)stream>>>(result, a, b, n);
    return hipGetLastError();
}

extern "C" int launch_vector_divide_kernel(
    float *result, const float *a, const float *b, int n,
    unsigned int grid_dim_x, unsigned int grid_dim_y, unsigned int grid_dim_z,
    unsigned int block_dim_x, unsigned int block_dim_y, unsigned int block_dim_z,
    unsigned int shared_mem_bytes, void *stream)
{
    dim3 grid(grid_dim_x, grid_dim_y, grid_dim_z);
    dim3 block(block_dim_x, block_dim_y, block_dim_z);
    
    vector_divide_kernel<<<grid, block, shared_mem_bytes, (hipStream_t)stream>>>(result, a, b, n);
    return hipGetLastError();
}

extern "C" int launch_vector_exp_kernel(
    float *result, const float *a, int n,
    unsigned int grid_dim_x, unsigned int grid_dim_y, unsigned int grid_dim_z,
    unsigned int block_dim_x, unsigned int block_dim_y, unsigned int block_dim_z,
    unsigned int shared_mem_bytes, void *stream)
{
    dim3 grid(grid_dim_x, grid_dim_y, grid_dim_z);
    dim3 block(block_dim_x, block_dim_y, block_dim_z);
    
    vector_exp_kernel<<<grid, block, shared_mem_bytes, (hipStream_t)stream>>>(result, a, n);
    return hipGetLastError();
}

extern "C" int launch_vector_log_kernel(
    float *result, const float *a, int n,
    unsigned int grid_dim_x, unsigned int grid_dim_y, unsigned int grid_dim_z,
    unsigned int block_dim_x, unsigned int block_dim_y, unsigned int block_dim_z,
    unsigned int shared_mem_bytes, void *stream)
{
    dim3 grid(grid_dim_x, grid_dim_y, grid_dim_z);
    dim3 block(block_dim_x, block_dim_y, block_dim_z);
    
    vector_log_kernel<<<grid, block, shared_mem_bytes, (hipStream_t)stream>>>(result, a, n);
    return hipGetLastError();
}

extern "C" int launch_vector_sqrt_kernel(
    float *result, const float *a, int n,
    unsigned int grid_dim_x, unsigned int grid_dim_y, unsigned int grid_dim_z,
    unsigned int block_dim_x, unsigned int block_dim_y, unsigned int block_dim_z,
    unsigned int shared_mem_bytes, void *stream)
{
    dim3 grid(grid_dim_x, grid_dim_y, grid_dim_z);
    dim3 block(block_dim_x, block_dim_y, block_dim_z);
    
    vector_sqrt_kernel<<<grid, block, shared_mem_bytes, (hipStream_t)stream>>>(result, a, n);
    return hipGetLastError();
}

extern "C" int launch_vector_pow_kernel(
    float *result, const float *a, float power, int n,
    unsigned int grid_dim_x, unsigned int grid_dim_y, unsigned int grid_dim_z,
    unsigned int block_dim_x, unsigned int block_dim_y, unsigned int block_dim_z,
    unsigned int shared_mem_bytes, void *stream)
{
    dim3 grid(grid_dim_x, grid_dim_y, grid_dim_z);
    dim3 block(block_dim_x, block_dim_y, block_dim_z);
    
    vector_pow_kernel<<<grid, block, shared_mem_bytes, (hipStream_t)stream>>>(result, a, power, n);
    return hipGetLastError();
}

extern "C" int launch_matrix_multiply_kernel(
    float *result, const float *a, const float *b, int m, int n, int k,
    unsigned int grid_dim_x, unsigned int grid_dim_y, unsigned int grid_dim_z,
    unsigned int block_dim_x, unsigned int block_dim_y, unsigned int block_dim_z,
    unsigned int shared_mem_bytes, void *stream)
{
    dim3 grid(grid_dim_x, grid_dim_y, grid_dim_z);
    dim3 block(block_dim_x, block_dim_y, block_dim_z);
    
    matrix_multiply_kernel<<<grid, block, shared_mem_bytes, (hipStream_t)stream>>>(result, a, b, m, n, k);
    return hipGetLastError();
}

extern "C" int launch_vector_reduce_sum_kernel(
    float* result, const float* input, int n,
    unsigned int grid_dim_x, unsigned int grid_dim_y, unsigned int grid_dim_z,
    unsigned int block_dim_x, unsigned int block_dim_y, unsigned int block_dim_z,
    unsigned int shared_mem_bytes, hipStream_t stream
) {
    if (n <= 0) {
        *result = 0.0f;
        return 0;
    }
    
    // Limit block size to avoid exceeding shared memory
    const unsigned int max_block_size = 1024;
    unsigned int block_size = block_dim_x * block_dim_y * block_dim_z;
    if (block_size > max_block_size) block_size = max_block_size;
    
    // Calculate grid size to cover the entire input
    unsigned int grid_size = (n + block_size - 1) / block_size;
    
    // Allocate temporary storage for block results
    float* d_block_sums = nullptr;
    hipError_t error = hipMalloc(&d_block_sums, grid_size * sizeof(float));
    if (error != hipSuccess) return (int)error;
    
    // Launch the first kernel to reduce blocks
    dim3 dimBlock(block_size, 1, 1);
    dim3 dimGrid(grid_size, 1, 1);
    unsigned int smem_size = block_size * sizeof(float);
    
    reduce_sum_kernel<<<dimGrid, dimBlock, smem_size, stream>>>(
        d_block_sums, input, n
    );
    
    // If grid_size is 1, we can directly copy the result
    if (grid_size == 1) {
        error = hipMemcpyAsync(result, d_block_sums, sizeof(float), 
                               hipMemcpyDeviceToDevice, stream);
        hipFree(d_block_sums);
        return (int)error;
    }
    
    // Otherwise, we need another reduction step
    // Reduce the block sums
    reduce_blocks_kernel<<<1, 1024, 1024 * sizeof(float), stream>>>(
        result, d_block_sums, grid_size
    );
    
    // Free temporary storage
    hipFree(d_block_sums);
    
    return 0;
}